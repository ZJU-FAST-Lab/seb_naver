#include "hip/hip_runtime.h"
#define checkCudaError( hipError_t ) __checkCudaError( hipError_t, __FILE__, __LINE__ )

#include <Eigen/Eigen>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <ctime>
#include <unistd.h>
#include <opencv2/photo/photo.hpp>
#include <opencv2/core/eigen.hpp>
#include <opencv2/opencv.hpp>
#include <cv_bridge/cv_bridge.h>

#define MAX_BLOCK 400
#define THREAD_NUM 256
#define CLOUD_NUM 65536
#define GRID_NUM 100000
#define SO2_NUM 36

using namespace Eigen;
using namespace std;

// Eigen::SelfAdjointEigenSolver<Eigen::Matrix<float, 6, 6>> esolver(ATA);
// Eigen::Matrix<float, 1, 6> matE;
// Eigen::Matrix<float, 6, 6> matV;
// matE = esolver.eigenvalues().real();
// matV = esolver.eigenvectors().real();

void __checkCudaError ( hipError_t result_t, const char * file, const int line )
{
    std::string error_string;

    if ( hipSuccess != result_t && hipErrorDeinitialized != result_t )
    {
        fprintf ( stderr, "\x1B[31m CUDA error encountered in file '%s', line %d\n Error %d: %s\n Terminating FIRE!\n \x1B[0m", file, line, result_t,
               hipGetErrorString ( result_t ) );
        printf("CUDA error encountered: %s", hipGetErrorString ( result_t ) );
        printf(". Terminating application.\n");
        throw std::runtime_error ( "checkCUDAError : ERROR: CUDA Error" );
    }
}

// map param
__constant__ int   size_pos[2];
__constant__ int   size_yaw;
__constant__ int   xy_num;
__constant__ int   xyyaw_num;
__constant__ int   inpaint_method;
__constant__ float length_pos[2];
__constant__ float length_yaw;
__constant__ float resolution_pos;
__constant__ float resolution_pos_inv;
__constant__ float resolution_yaw;
__constant__ float resolution_yaw_inv;
__constant__ float origin[2];
__constant__ float min_var;
__constant__ float max_var;
__constant__ float hori_var;
__constant__ float mahalanobis_threshold;
__constant__ float normal_radius;
__constant__ float min_cosxi;
__constant__ float max_curvature;
__constant__ float weight_xi;
__constant__ float weight_curvature;
__constant__ float ignore_z_min;
__constant__ float ignore_z_max;
__constant__ float beam_sigma2;
__constant__ float body2sensor_T[3];

// map data
__device__ bool map_nan[GRID_NUM];
__device__ float elevation[GRID_NUM];
__device__ float var[GRID_NUM];
__device__ float inpainted[GRID_NUM];
__device__ float smooth[GRID_NUM];
__device__ float normal_x[GRID_NUM];
__device__ float normal_y[GRID_NUM];
__device__ float risk[GRID_NUM];
__device__ float sdf[GRID_NUM];

// cloud temp
__device__ int temp_idx[CLOUD_NUM];
__device__ float temp_var[CLOUD_NUM];
__device__ float temp_height[CLOUD_NUM];

__host__ __device__ void wrapIndexOne(int& idx, int buffer_size)
{
    if (idx < buffer_size)
    {
        if(idx >= 0)
            return;
        else if(idx >= -buffer_size)
        {
            idx += buffer_size;
            return;
        }
        else
        {
            idx = idx % buffer_size;
            idx += buffer_size;
        }
    }
    else if(idx < buffer_size*2)
    {
        idx -= buffer_size;
        return;
    }
    else
        idx = idx % buffer_size;
}

__device__ int signFunc(const int val)
{
    return static_cast<int>(0 < val) - static_cast<int>(val < 0);
} 

__device__ void boundPos(Vector2f& pos, const Vector2f& position)
{
    Vector2f positionShifted = pos - position;
    positionShifted[0] += origin[0];
    positionShifted[1] += origin[1];

    for (int i = 0; i < positionShifted.size(); i++)
    {
        float epsilon = 10.0 * 1e-10;
        if (fabs(pos(i)) > 1.0)
            epsilon *= fabs(pos(i));

        if (positionShifted(i) <= 0)
        {
            positionShifted(i) = epsilon;
            continue;
        }
        if (positionShifted(i) >= length_pos[i])
        {
            positionShifted(i) = length_pos[i] - epsilon;
            continue;
        }
    }

    pos = positionShifted + position;
    pos[0] -= origin[0];
    pos[1] -= origin[1];
}


__device__ void computerEigenvalue(float *pMatrix, int nDim, float *maxvector, float *curvature, float dbEps, int nJt)
{
    float pdblVects[9];
    float pdbEigenValues[3];
    
	for(int i = 0; i < nDim; i ++) 
	{   
		pdblVects[i*nDim+i] = 1.0f; 
		for(int j = 0; j < nDim; j ++) 
		{ 
			if(i != j)   
				pdblVects[i*nDim+j]=0.0f; 
		} 
	} 
 
	int nCount = 0;
	while(1)
	{
		float dbMax = pMatrix[1];
		int nRow = 0;
		int nCol = 1;
		for (int i = 0; i < nDim; i ++)
		{
			for (int j = 0; j < nDim; j ++)
			{
				float d = fabs(pMatrix[i*nDim+j]); 
 
				if((i!=j) && (d> dbMax)) 
				{ 
					dbMax = d;   
					nRow = i;   
					nCol = j; 
				} 
			}
		}
 
		if(dbMax < dbEps) 
			break;  
 
		if(nCount > nJt)
			break;
 
		nCount++;
 
		float dbApp = pMatrix[nRow*nDim+nRow];
		float dbApq = pMatrix[nRow*nDim+nCol];
		float dbAqq = pMatrix[nCol*nDim+nCol];
 
		float dbAngle = 0.5*atan2(-2*dbApq,dbAqq-dbApp);
		float dbSinTheta = sin(dbAngle);
		float dbCosTheta = cos(dbAngle);
		float dbSin2Theta = sin(2*dbAngle);
		float dbCos2Theta = cos(2*dbAngle);
 
		pMatrix[nRow*nDim+nRow] = dbApp*dbCosTheta*dbCosTheta + 
			dbAqq*dbSinTheta*dbSinTheta + 2*dbApq*dbCosTheta*dbSinTheta;
		pMatrix[nCol*nDim+nCol] = dbApp*dbSinTheta*dbSinTheta + 
			dbAqq*dbCosTheta*dbCosTheta - 2*dbApq*dbCosTheta*dbSinTheta;
		pMatrix[nRow*nDim+nCol] = 0.5*(dbAqq-dbApp)*dbSin2Theta + dbApq*dbCos2Theta;
		pMatrix[nCol*nDim+nRow] = pMatrix[nRow*nDim+nCol];
 
		for(int i = 0; i < nDim; i ++) 
		{ 
			if((i!=nCol) && (i!=nRow)) 
			{ 
				int u = i*nDim + nRow;	//p  
				int w = i*nDim + nCol;	//q
				dbMax = pMatrix[u]; 
				pMatrix[u]= pMatrix[w]*dbSinTheta + dbMax*dbCosTheta; 
				pMatrix[w]= pMatrix[w]*dbCosTheta - dbMax*dbSinTheta; 
			} 
		} 
 
		for (int j = 0; j < nDim; j ++)
		{
			if((j!=nCol) && (j!=nRow)) 
			{ 
				int u = nRow*nDim + j;	//p
				int w = nCol*nDim + j;	//q
				dbMax = pMatrix[u]; 
				pMatrix[u]= pMatrix[w]*dbSinTheta + dbMax*dbCosTheta; 
				pMatrix[w]= pMatrix[w]*dbCosTheta - dbMax*dbSinTheta; 
			} 
		}
 
		for(int i = 0; i < nDim; i ++) 
		{ 
			int u = i*nDim + nRow;		//p   
			int w = i*nDim + nCol;		//q
			dbMax = pdblVects[u]; 
			pdblVects[u] = pdblVects[w]*dbSinTheta + dbMax*dbCosTheta; 
			pdblVects[w] = pdblVects[w]*dbCosTheta - dbMax*dbSinTheta; 
		} 
 
	}
    
    int min_id = 0;
	float minEigenvalue;
    float sumEigenvalue = 0.0;

	for(int i = 0; i < nDim; i ++) 
	{   
		pdbEigenValues[i] = pMatrix[i*nDim+i];
        sumEigenvalue += pdbEigenValues[i];
        if(i == 0)
            minEigenvalue = pdbEigenValues[i];
        else
        {
            if(minEigenvalue > pdbEigenValues[i])
            {
                minEigenvalue = pdbEigenValues[i];
                min_id = i;	
            }
        }
    } 

    for(int i = 0; i < nDim; i ++) 
    {  
        maxvector[i] = pdblVects[min_id + nDim * i];
    }

    *curvature = 3.0 * minEigenvalue / sumEigenvalue;
}

__global__ void init_map()
{
    int map_idx = threadIdx.x + blockIdx.x * blockDim.x;

    if (map_idx < xy_num)
    {
        map_nan[map_idx] = true;
        sdf[map_idx] = 0.0;
    }

    //! xulong
    // if (map_idx == 0)
    // {
    //     printf("device params:\n");
    //     printf("size_pos[0] = %d\n", size_pos[0]);
    //     printf("size_pos[1] = %d\n", size_pos[1]);
    //     printf("size_yaw = %d\n", size_yaw);
    //     printf("xy_num = %d\n", xy_num);
    //     printf("xyyaw_num = %d\n", xyyaw_num);
    //     printf("resolution_pos_inv = %f\n", resolution_pos_inv);
    //     printf("resolution_yaw_inv = %f\n", resolution_yaw_inv);
    //     printf("resolution_pos = %f\n", resolution_pos);
    //     printf("resolution_yaw = %f\n", resolution_yaw);
    //     printf("length_pos[0] = %f\n", length_pos[0]);
    //     printf("length_pos[1] = %f\n", length_pos[1]);
    //     printf("length_yaw =6.284\n");
    //     printf("origin[0] = %f\n", origin[0]);
    //     printf("origin[1] = %f\n", origin[1]);
    //     printf("min_var = %f\n", min_var);
    //     printf("max_var = %f\n", max_var);
    //     printf("hori_var = %f\n", hori_var);
    //     printf("mahalanobis_threshold, %f\n", mahalanobis_threshold);
    //     printf("normal_radius = %f\n", normal_radius);
    //     printf("min_cosxi = %f\n", min_cosxi);
    //     printf("max_curvature = %f\n", max_curvature);
    //     printf("weight_xi = %f\n", weight_xi);
    //     printf("weight_curvature = %f\n", weight_curvature);
    //     printf("ignore_z_min = %f\n", ignore_z_min);
    //     printf("ignore_z_max = %f\n", ignore_z_max);
    //     printf("body2sensor_T[0] = %f\n", body2sensor_T[0]);
    //     printf("body2sensor_T[1] = %f\n", body2sensor_T[1]);
    //     printf("body2sensor_T[2] = %f\n", body2sensor_T[2]);
    //     printf("beam_sigma2 = %f\n", beam_sigma2);
    // }
}

__global__ void clear_map(Vector2i clear_idx, 
                        Vector2i clear_nums, 
                        Vector2i clear_idx2,
                        Vector2i clear_nums2,
                        bool clear_all,
                        bool clear_two)
{
    int map_idx = threadIdx.x + blockIdx.x * blockDim.x;
    int size_row = size_pos[0];
    int size_col = size_pos[1];

    if (clear_all)
    {
        if (map_idx < xy_num)
            map_nan[map_idx] = true;
    }

    else if (map_idx < xy_num)
    {
        int clear_rows = clear_nums[0];
        int clear_cols = clear_nums[1];
        int cell_num_row = size_col * clear_rows;
        int cell_num_col = size_row * clear_cols;
        if (map_idx < cell_num_row)
            map_nan[map_idx / clear_rows * size_row + map_idx % clear_rows + clear_idx[0]] = true;
        if (map_idx < cell_num_col)
            map_nan[clear_idx[1] * size_row + map_idx] = true;
        if (clear_two)
        {
            int clear_rows2 = clear_nums2[0];
            int clear_cols2 = clear_nums2[1];
            int cell_num_row2 = size_col * clear_rows2;
            int cell_num_col2 = size_row * clear_cols2;
            if (map_idx < cell_num_row2)
                map_nan[map_idx / clear_rows2 * size_row + map_idx % clear_rows2 + clear_idx2[0]] = true;
            if (map_idx < cell_num_col2)
                map_nan[clear_idx2[1] * size_row + map_idx] = true;
        }
    }
}

__global__ void compute_var(int point_num,
                            float* dev_points, 
                            Vector3f world_T,
                            Matrix3f body_R,
                            Matrix3f body_Cov_R,
                            float body_Cov_Z,
                            Array2i start_index,
                            Vector2f position)
{
    // deal with cloud var
    const int pt_num = point_num;
    const float ignore_z_min_ = ignore_z_min;
    const float ignore_z_max_ = ignore_z_max;
    const float beam_sigma2_ = beam_sigma2;
    const float resolution_pos_inv_ = resolution_pos_inv;

    // check if in num range
    int pt_idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (pt_idx < pt_num)
    {
        Vector3f point_world(dev_points[pt_idx*3], dev_points[pt_idx*3+1], dev_points[pt_idx*3+2]);
        Map<Vector2f> origin_(origin);

        // check if in map range
        Vector2f pos_map = origin_ + position - point_world.head(2);
        if (pos_map.x() >= 0.0 && pos_map.y() >= 0.0 && \
            pos_map.x() < length_pos[0] && pos_map.y() < length_pos[1])
        {
            // check if in height range
            float height_body = body_R.col(2).dot(point_world) + world_T[2];
            if (height_body > ignore_z_min_ && height_body < ignore_z_max_)
            {
                // compute var
                Vector3f point_sensor;
                point_sensor[0] = body_R.col(0).dot(point_world) + world_T[0] + body2sensor_T[0];
                point_sensor[1] = body_R.col(1).dot(point_world) + world_T[1] + body2sensor_T[1];
                point_sensor[2] = height_body + body2sensor_T[2];

                Vector3f Js = body_R.row(2);
                Vector3f Jr = Vector3f::Zero();
                Vector3f SigmaS = Vector3f::Ones();

                Vector3f point_sensor2 = point_sensor.cwiseAbs2();
                SigmaS = point_sensor2 / point_sensor2.sum() * beam_sigma2_;

                Vector3f dVec = point_sensor;
                dVec[0] -= body2sensor_T[0];
                dVec[1] -= body2sensor_T[1];
                dVec[2] -= body2sensor_T[2];
                Jr[0] = -dVec[2] * Js[1] + dVec[1] * Js[2];
                Jr[1] = dVec[2] * Js[0] - dVec[0] * Js[2];
                Jr[2] = -dVec[1] * Js[0] + dVec[0] * Js[1];
                Vector3f sJr;
                sJr[0] = body_Cov_R.row(0).dot(Jr);
                sJr[1] = body_Cov_R.row(1).dot(Jr);
                sJr[2] = body_Cov_R.row(2).dot(Jr);
                float variance = body_Cov_Z
                                + (SigmaS.cwiseProduct(Js)).dot(Js)
                                + sJr.dot(Jr);

                // add to temp
                Array2i index;
                Array2f indexf = pos_map.array() * resolution_pos_inv_;
                index(0) = (int)indexf(0);
                index(1) = (int)indexf(1);
                if ( start_index[0] != 0 || start_index[1] != 0)
                {
                    index[0] += start_index[0];
                    index[1] += start_index[1];
                    for (int i=0; i<2; i++)
                        wrapIndexOne(index[i], size_pos[i]);
                }
                int true_idx = index[1] * size_pos[0] + index[0];
                temp_height[pt_idx] = point_world.z();
                temp_idx[pt_idx] = true_idx;
                temp_var[pt_idx] = variance;
            }
            else
                temp_idx[pt_idx] = -1;
        }
        else
            temp_idx[pt_idx] = -1;
    }
}

__global__ void compute_elevation(int point_num)
{
    // add to map ( var update using KF )
    int map_idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (map_idx < xy_num)
    {
        sdf[map_idx] = 0.0;
        risk[map_idx] = 0.0;
        for (int i=0; i<point_num; i++)
        {
            int true_idx = temp_idx[i];
            float height = temp_height[i];
            float tvar = temp_var[i];
            if (true_idx == -1 || true_idx != map_idx)
                continue;

            if (map_nan[map_idx])
            {
                elevation[map_idx] = height;
                var[map_idx] = tvar;
                map_nan[map_idx] = false;
            }else
            {
                float mahalanobisDistance = fabs(height - elevation[map_idx]) / sqrt(tvar);
                if (mahalanobisDistance > mahalanobis_threshold)
                {
                    elevation[map_idx] = height;
                    var[map_idx] = tvar;
                }
                else
                {
                    float var_old = var[map_idx];
                    float ele_old = elevation[map_idx];
                    elevation[map_idx] = (var_old * height + tvar * ele_old)
                                                        / (var_old + tvar);
                    var[map_idx] = (var_old * tvar) / (var_old + tvar);
                }
            }
            var[map_idx] = max(min_var, var[map_idx]);
        }
        if (map_nan[map_idx])
        {
            elevation[map_idx] = NAN;
            var[map_idx] = NAN;
        }
    }
}

__global__ void compute_inpaint_spiral(Array2i start_index,
                                        Vector2f position)
{
    int map_idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (map_idx < xy_num)
    {
        int map_idx_true = map_idx / size_pos[1] + (map_idx % size_pos[1]) * size_pos[0];
        if (inpaint_method != 1)
        {
            inpainted[map_idx_true] = elevation[map_idx_true];
            return;
        }
        if (map_nan[map_idx_true])
        {
            inpainted[map_idx_true] = NAN;
            Array2i idx(map_idx/size_pos[1], map_idx%size_pos[1]);
            if ( start_index[0] != 0 || start_index[1] != 0)
            {
                idx[0] -= start_index[0];
                idx[1] -= start_index[1];
                for (int i=0; i<2; i++)
                    wrapIndexOne(idx[i], size_pos[i]);
            }

            float radius2 = length_pos[0] * length_pos[0] + length_pos[1] * length_pos[1];
            int n_rings = (int) (sqrt(radius2) / resolution_pos) + 1;
            int distance_ = 0;
            do
            {
                bool get_nearest = false;
                distance_++;
                Array2i point(distance_, 0);
                Array2i pointInMap;
                Array2i normal;
                do
                {
                    pointInMap = point + idx;
                    if (pointInMap[0] >= 0 && pointInMap[1] >= 0 &&
                        pointInMap[0] < size_pos[0] && pointInMap[1] < size_pos[1])
                    {
                         pointInMap += start_index;
                        for (int i=0; i<2; i++)
                            wrapIndexOne(pointInMap[i], size_pos[i]);
                        // col-major
                        int true_idx = pointInMap[1] * size_pos[0] + pointInMap[0];
                        if (!map_nan[true_idx])
                        {
                            get_nearest = true;
                            inpainted[map_idx_true] = elevation[true_idx];
                            break;
                        }
                    }

                    normal[0] = -signFunc(point[1]);
                    normal[1] = signFunc(point[0]);
                    if (normal[0] !=0 && (int)(Vector2f(point[0] + normal[0], point[1]).norm()) == distance_)
                        point[0] += normal[0];
                    if (normal[0] !=0 && (int)(Vector2f(point[0], point[1] + normal[1]).norm()) == distance_)
                        point[1] += normal[1];
                    else
                    {
                        point += normal;
                    }
                } while (point[0]!=distance_ || point[1]!=0);
                if (get_nearest)
                    break;
            }while(distance_<n_rings);
        }
        else
        {
            inpainted[map_idx_true] = elevation[map_idx_true];
        }
    }
}

__global__ void compute_map_r2(Array2i start_index,
                                Vector2f position)
{
    // normal (circle iterator)
    int map_idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (map_idx < xy_num)
    {
        // for col-major
        int map_idx_true = map_idx / size_pos[1] + (map_idx % size_pos[1]) * size_pos[0];
        if (!map_nan[map_idx_true] || inpaint_method!=0)
        {
            Vector2i idx(map_idx/size_pos[1], map_idx%size_pos[1]);
            Vector2f origin_off(origin[0], origin[1]);
            origin_off[0] -= 0.5 * resolution_pos;
            origin_off[1] -= 0.5 * resolution_pos;
            if ( start_index[0] != 0 || start_index[1] != 0)
            {
                idx[0] -= start_index[0];
                idx[1] -= start_index[1];
                for (int i=0; i<2; i++)
                    wrapIndexOne(idx[i], size_pos[i]);
            }
            Vector2f center = position + origin_off - (idx.cast<float>() * resolution_pos).matrix();
            Vector2f top_left = center.array() + normal_radius;
            Vector2f bottom_right = center.array() - normal_radius;
            boundPos(top_left, position);
            boundPos(bottom_right, position);
            Array2i sub_start;
            sub_start[0] = (int) ( (origin[0] + position[0] - top_left[0]) * resolution_pos_inv );
            sub_start[1] = (int) ( (origin[1] + position[1] - top_left[1]) * resolution_pos_inv );
            Array2i sub_end;
            sub_end[0] = (int) ( (origin[0] + position[0] - bottom_right[0]) * resolution_pos_inv );
            sub_end[1] = (int) ( (origin[1] + position[1] - bottom_right[1]) * resolution_pos_inv );
            if ( start_index[0] != 0 || start_index[1] != 0)
            {
                sub_start[0] += start_index[0];
                sub_start[1] += start_index[1];
                for (int i=0; i<2; i++)
                    wrapIndexOne(sub_start[i], size_pos[i]);
                sub_end[0] += start_index[0];
                sub_end[1] += start_index[1];
                for (int i=0; i<2; i++)
                    wrapIndexOne(sub_end[i], size_pos[i]);
                
                sub_start[0] -= start_index[0];
                sub_start[1] -= start_index[1];
                for (int i=0; i<2; i++)
                    wrapIndexOne(sub_start[i], size_pos[i]);
                sub_end[0] -= start_index[0];
                sub_end[1] -= start_index[1];
                for (int i=0; i<2; i++)
                    wrapIndexOne(sub_end[i], size_pos[i]);
            }
            Array2i buffer_size = sub_end - sub_start + Array2i::Ones();
            Vector3f mean_p = Vector3f::Zero();
            int cnt_p = 0;
            Vector3f temp_poses[60];
            for (int i=0; i<buffer_size[0]; i++)
            {
                for (int j=0; j<buffer_size[1]; j++)
                {
                    Array2i tempIndex = sub_start + Array2i(i, j);
                    if ( start_index[0] != 0 || start_index[1] != 0)
                    {
                        tempIndex += start_index;
                        for (int i = 0; i < tempIndex.size(); i++)
                            wrapIndexOne(tempIndex(i), size_pos[i]);
                    }
                    
                    // col-major
                    int true_idx = tempIndex[1] * size_pos[0] + tempIndex[0];
                    if (!map_nan[true_idx] || inpaint_method!=0)
                    {
                        if ( start_index[0] != 0 || start_index[1] != 0)
                        {
                            tempIndex[0] -= start_index[0];
                            tempIndex[1] -= start_index[1];
                            for (int i=0; i<2; i++)
                                wrapIndexOne(tempIndex[i], size_pos[i]);
                        }
                        Vector2f temp_pos = position + origin_off - (tempIndex.cast<float>() * resolution_pos).matrix();
                        if ((temp_pos - center).norm() < normal_radius)
                        {
                            Vector3f temp_pos3;
                            temp_pos3.head(2) = temp_pos;
                            temp_pos3[2] = inpainted[true_idx];
                            mean_p[0] += temp_pos[0];
                            mean_p[1] += temp_pos[1];
                            mean_p[2] += temp_pos3[2];
                            temp_poses[cnt_p] = temp_pos3;
                            cnt_p ++;
                        }
                    }
                }
            }

            mean_p[0] = mean_p[0] / (float)cnt_p;
            mean_p[1] = mean_p[1] / (float)cnt_p;
            mean_p[2] = mean_p[2] / (float)cnt_p;
            
            smooth[map_idx_true] = mean_p[2];
            if (cnt_p > 7)
            {
                float pMatrix[9] = {0};
                for(int i = 0; i < cnt_p; i ++)
                {
                    pMatrix[0] = pMatrix[0] + (temp_poses[i][0] - mean_p[0]) * (temp_poses[i][0] - mean_p[0]);
                    pMatrix[4] = pMatrix[4] + (temp_poses[i][1] - mean_p[1]) * (temp_poses[i][1] - mean_p[1]);
                    pMatrix[8] = pMatrix[8] + (temp_poses[i][2] - mean_p[2]) * (temp_poses[i][2] - mean_p[2]);
                    pMatrix[1] = pMatrix[1] + (temp_poses[i][0] - mean_p[0]) * (temp_poses[i][1] - mean_p[1]);
                    pMatrix[2] = pMatrix[2] + (temp_poses[i][0] - mean_p[0]) * (temp_poses[i][2] - mean_p[2]);
                    pMatrix[5] = pMatrix[5] + (temp_poses[i][1] - mean_p[1]) * (temp_poses[i][2] - mean_p[2]);
                    pMatrix[3] = pMatrix[1];
                    pMatrix[6] = pMatrix[2];
                    pMatrix[7] = pMatrix[5];
                }
                
                float dbEps = 0.01;
                int nJt = 30;
                float normal_vec[3];
                float curvature;
                computerEigenvalue(pMatrix, 3, normal_vec, &curvature, dbEps, nJt);
                
                float cos_xi;
                if (normal_vec[2] > 0)
                {
                    normal_x[map_idx_true] = normal_vec[0];
                    normal_y[map_idx_true] = normal_vec[1];
                    cos_xi = normal_vec[2];
                }
                else
                {
                    normal_x[map_idx_true] = -normal_vec[0];
                    normal_y[map_idx_true] = -normal_vec[1];
                    cos_xi = -normal_vec[2];
                }

                if (cos_xi < min_cosxi || curvature > max_curvature)
                {
                    risk[map_idx_true] = 1.0;
                    sdf[map_idx_true] = 1.0;
                }
                else
                {
                    risk[map_idx_true] = weight_xi * (1.0 - cos_xi) / (1.0 - min_cosxi) + weight_curvature * curvature;
                    sdf[map_idx_true] = 0.0;
                }
            }
            else
            {
                risk[map_idx_true] = 0.0;
                sdf[map_idx_true] = 0.0;
                normal_x[map_idx_true] = 0.0;
                normal_y[map_idx_true] = 0.0;
            }
        }

        // aggresive
        if (map_nan[map_idx_true])
        {
            risk[map_idx_true] = 0.0;
            sdf[map_idx_true] = 0.0;
        }
    }
}

__global__ void compute_sdf(Array2i start_index,
                            Vector2f position,
                            int dim)
{
    ;
}

void gpuInit(const Vector2i& size_pos_,
            int   size_yaw_,
            const Vector2f& length_pos_,
            float resolution_pos_,
            float resolution_yaw_,
            const Vector2f& position_,
            float min_var_,
            float max_var_,
            float hori_var_,
            float mahalanobis_threshold_,
            float normal_radius_,
            float min_cosxi_,
            float max_curvature_,
            float weight_xi_,
            float weight_curvature_,
            float ignore_z_min_,
            float ignore_z_max_,
            float beam_sigma2_,
            const Vector3f& body2sensor_T_)
{
    // set up
    hipSetDevice(0);
    bool h_map_nan[GRID_NUM];
    float h_elevation[GRID_NUM];
    float h_var[GRID_NUM];
    float h_inpainted[GRID_NUM];
    float h_smooth[GRID_NUM];
    float h_normal_x[GRID_NUM];
    float h_normal_y[GRID_NUM];
    float h_risk[GRID_NUM];
    float h_sdf[GRID_NUM];

    int h_temp_idx[CLOUD_NUM];
    float h_temp_var[CLOUD_NUM];
    float h_temp_height[CLOUD_NUM];

    checkCudaError(hipMemcpyToSymbol(HIP_SYMBOL(map_nan), &h_map_nan, sizeof(bool)*GRID_NUM, 0, hipMemcpyHostToDevice));
    checkCudaError(hipMemcpyToSymbol(HIP_SYMBOL(elevation), &h_elevation, sizeof(float)*GRID_NUM, 0, hipMemcpyHostToDevice));
    checkCudaError(hipMemcpyToSymbol(HIP_SYMBOL(var), &h_var, sizeof(float)*GRID_NUM, 0, hipMemcpyHostToDevice));
    checkCudaError(hipMemcpyToSymbol(HIP_SYMBOL(inpainted), &h_inpainted, sizeof(float)*GRID_NUM, 0, hipMemcpyHostToDevice));
    checkCudaError(hipMemcpyToSymbol(HIP_SYMBOL(smooth), &h_smooth, sizeof(float)*GRID_NUM, 0, hipMemcpyHostToDevice));
    checkCudaError(hipMemcpyToSymbol(HIP_SYMBOL(normal_x), &h_normal_x, sizeof(float)*GRID_NUM, 0, hipMemcpyHostToDevice));
    checkCudaError(hipMemcpyToSymbol(HIP_SYMBOL(normal_y), &h_normal_y, sizeof(float)*GRID_NUM, 0, hipMemcpyHostToDevice));
    checkCudaError(hipMemcpyToSymbol(HIP_SYMBOL(risk), &h_risk, sizeof(float)*GRID_NUM, 0, hipMemcpyHostToDevice));
    checkCudaError(hipMemcpyToSymbol(HIP_SYMBOL(sdf), &h_sdf, sizeof(float)*GRID_NUM, 0, hipMemcpyHostToDevice));
    checkCudaError(hipMemcpyToSymbol(HIP_SYMBOL(temp_idx), &h_temp_idx, sizeof(int)*CLOUD_NUM, 0, hipMemcpyHostToDevice));
    checkCudaError(hipMemcpyToSymbol(HIP_SYMBOL(temp_var), &h_temp_var, sizeof(float)*CLOUD_NUM, 0, hipMemcpyHostToDevice));
    checkCudaError(hipMemcpyToSymbol(HIP_SYMBOL(temp_height), &h_temp_height, sizeof(float)*CLOUD_NUM, 0, hipMemcpyHostToDevice));
    
    // get params
    int xy_num_ = size_pos_[0] * size_pos_[1];
    int xyyaw_num_ = xy_num_ * size_yaw_;
    float pix2 = 6.284f;
    float res_pos_inv_ = 1.0f / resolution_pos_;
    float res_yaw_inv_ = 1.0f / resolution_yaw_;
    float origin_[2];
    origin_[0] = 0.5 * length_pos_[0];
    origin_[1] = 0.5 * length_pos_[1];
    hipMemcpyToSymbol(HIP_SYMBOL(size_pos), &size_pos_, sizeof(int) * 2);
    hipMemcpyToSymbol(HIP_SYMBOL(size_yaw), &size_yaw_, sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL(xy_num), &xy_num_, sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL(xyyaw_num), &xyyaw_num_, sizeof(int));
    hipMemcpyToSymbol(HIP_SYMBOL(resolution_pos_inv), &res_pos_inv_, sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(resolution_yaw_inv), &res_yaw_inv_, sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(resolution_pos), &resolution_pos_, sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(resolution_yaw), &resolution_yaw_, sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(length_pos), &length_pos_, sizeof(float) * 2);
    hipMemcpyToSymbol(HIP_SYMBOL(length_yaw), &pix2, sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(origin), &origin_, sizeof(float) * 2);
    hipMemcpyToSymbol(HIP_SYMBOL(min_var), &min_var_, sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(max_var), &max_var_, sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(hori_var), &hori_var_, sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(mahalanobis_threshold), &mahalanobis_threshold_, sizeof(float)); 
    hipMemcpyToSymbol(HIP_SYMBOL(normal_radius), &normal_radius_, sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(min_cosxi), &min_cosxi_, sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(max_curvature), &max_curvature_, sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(weight_xi), &weight_xi_, sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(weight_curvature), &weight_curvature_, sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(ignore_z_min), &ignore_z_min_, sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(ignore_z_max), &ignore_z_max_, sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(body2sensor_T), &body2sensor_T_, sizeof(float) * 3);
    hipMemcpyToSymbol(HIP_SYMBOL(beam_sigma2), &beam_sigma2_, sizeof(float));
    int inpaint_method_ = 1;
    hipMemcpyToSymbol(HIP_SYMBOL(inpaint_method), &inpaint_method_, sizeof(float));

    int num_block =(xy_num_ + THREAD_NUM - 1) / THREAD_NUM;
	init_map<<<num_block, THREAD_NUM>>>();
}

void computerMapR2(const int& point_num_,
                    float* points_world,
                    const float& resolution_pos_,
                    const Vector2i& size_pos_,
                    const Vector3f& body_T_,
                    const Matrix3f& body_R_,
                    const Matrix4f& body_Cov_ZR_,
                    MatrixXf& elevation_,
                    MatrixXf& var_,
                    MatrixXf& inpainted_,
                    MatrixXf& smooth_,
                    MatrixXf& normal_x_,
                    MatrixXf& normal_y_,
                    MatrixXf& risk_,
                    MatrixXf& sdf_,
                    Array2i& start_idx_,
                    Vector2f& position_,
                    bool verbose_)
{
    // move map
    Array2i indexShift;
    Vector2f positionShift = body_T_.head(2) - position_;
    
    Vector2f positionShiftTemp = (positionShift.array() / resolution_pos_).matrix();
    for (int i = 0; i < indexShift.size(); i++)
        indexShift[i] = static_cast<int>(positionShiftTemp[i] + 0.5 * (positionShiftTemp[i] > 0 ? 1 : -1));

    Vector2f alignedPositionShift = (indexShift.cast<float>() * resolution_pos_).matrix();
    indexShift = -indexShift;

    // Delete fields that fall out of map (and become empty cells).
    bool clear_all_ = false;
    bool clear_two_ = false;
    Vector2i clear_idx_ = Vector2i::Zero();
    Vector2i clear_nums_ = Vector2i::Zero();
    Vector2i clear_idx2_ = Vector2i::Zero();
    Vector2i clear_nums2_ = Vector2i::Zero();
    
    for (int i = 0; i < indexShift.size(); i++)
    {
        if (indexShift(i) != 0)
        {
            if (abs(indexShift(i)) >= size_pos_[i])
                clear_all_ = true;
            else
            {
                // Drop cells out of map.
                int sign = (indexShift(i) > 0 ? 1 : -1);
                int startIndex = start_idx_[i] - (sign < 0 ? 1 : 0);
                int endIndex = startIndex - sign + indexShift(i);
                int nCells = abs(indexShift(i));
                int index = (sign > 0 ? startIndex : endIndex);
                wrapIndexOne(index, size_pos_[i]);

                if (index + nCells <= size_pos_[i])
                {
                    // One region to drop.
                    clear_idx_[i] = index;
                    clear_nums_[i] = nCells;
                }
                else
                {
                    // Two regions to drop.
                    clear_two_ = true;
                    int firstIndex = index;
                    int firstNCells = size_pos_[i] - firstIndex;
                    clear_idx_[i] = firstIndex;
                    clear_nums_[i] = firstNCells;

                    int secondIndex = 0;
                    int secondNCells = nCells - firstNCells;
                    clear_idx2_[i] = secondIndex;
                    clear_nums2_[i] = secondNCells;
                }
            }
        }
    }
    
    // clear map
    int xy_num_ = size_pos_[0]*size_pos_[1];
    int num_block =(xy_num_ + THREAD_NUM - 1) / THREAD_NUM;
	clear_map<<<num_block, THREAD_NUM>>>(clear_idx_, 
                                        clear_nums_, 
                                        clear_idx2_,
                                        clear_nums2_,
                                        clear_all_,
                                        clear_two_);

    // update information.
    start_idx_[0] += indexShift[0];
    start_idx_[1] += indexShift[1];
    wrapIndexOne(start_idx_[0], size_pos_[0]);
    wrapIndexOne(start_idx_[1], size_pos_[1]);
    position_[0] += alignedPositionShift[0];
    position_[1] += alignedPositionShift[1];

    // get points
    Vector3f world_T_ = -body_R_.transpose() * body_T_;
    float* dev_points;
    hipMalloc((void**)&dev_points, point_num_ * sizeof(float) * 3);
    hipMemcpy(dev_points, points_world, point_num_ * sizeof(float) * 3, hipMemcpyHostToDevice);

    // compute var
    int blocksPerGrid =(point_num_ + THREAD_NUM - 1) / THREAD_NUM;
	compute_var<<<blocksPerGrid, THREAD_NUM>>>(point_num_,
                                                dev_points, 
                                                world_T_,
                                                body_R_,
                                                body_Cov_ZR_.bottomRightCorner(3, 3),
                                                body_Cov_ZR_(0, 0),
                                                start_idx_,
                                                position_
                                                );
    // usleep(1);
    hipDeviceSynchronize();
    hipFree(dev_points);

    // compute map
    blocksPerGrid =(xy_num_ + THREAD_NUM - 1) / THREAD_NUM;
	compute_elevation<<<blocksPerGrid, THREAD_NUM>>>(point_num_);
    // usleep(1);
    hipDeviceSynchronize();

    // inpaint map
    compute_inpaint_spiral<<<blocksPerGrid, THREAD_NUM>>>(start_idx_, position_);
    hipDeviceSynchronize();

    // compute r2 normal and cost
    compute_map_r2<<<blocksPerGrid, THREAD_NUM>>>(start_idx_, position_);
    hipDeviceSynchronize();

    // get map
    if (verbose_)
    {
        hipMemcpyFromSymbol(elevation_.data(), elevation, sizeof(float)*xy_num_);
        hipMemcpyFromSymbol(var_.data(), var, sizeof(float)*xy_num_);
        hipMemcpyFromSymbol(inpainted_.data(), inpainted, sizeof(float)*xy_num_);
        hipMemcpyFromSymbol(smooth_.data(), smooth, sizeof(float)*xy_num_);
        hipMemcpyFromSymbol(normal_x_.data(), normal_x, sizeof(float)*xy_num_);
        hipMemcpyFromSymbol(normal_y_.data(), normal_y, sizeof(float)*xy_num_);
        hipMemcpyFromSymbol(risk_.data(), risk, sizeof(float)*xy_num_);
    }
    hipMemcpyFromSymbol(sdf_.data(), sdf, sizeof(float)*xy_num_);
    hipDeviceSynchronize();
}
